
#include <hip/hip_runtime.h>
#define MAX_WORD_LENGTH 128

extern "C" __global__ void compute_edit_distance(const char* word, int word_len,
                                                 const char* word_list,
                                                 int* word_lengths,
                                                 int* distances,
                                                 int num_words) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_words) return;

    const char* current_word = word_list + idx * MAX_WORD_LENGTH;
    int current_len = word_lengths[idx];

    // Allocate local memory for the DP table
    int dp[MAX_WORD_LENGTH + 1][MAX_WORD_LENGTH + 1];

    // Initialize the DP table
    for (int i = 0; i <= word_len; ++i) dp[i][0] = i;
    for (int j = 0; j <= current_len; ++j) dp[0][j] = j;

    // Compute the edit distance using dynamic programming
    for (int i = 1; i <= word_len; ++i) {
        for (int j = 1; j <= current_len; ++j) {
            if (word[i - 1] == current_word[j - 1]) {
                dp[i][j] = dp[i - 1][j - 1];
            } else {
                dp[i][j] =
                    1 + min(dp[i - 1][j], min(dp[i][j - 1], dp[i - 1][j - 1]));
            }
        }
    }

    // Store the result
    distances[idx] = dp[word_len][current_len];
}
